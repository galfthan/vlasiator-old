#include "hip/hip_runtime.h"
/*
 *  Anything that needs to be translated by nvcc goes here.
 */

#include "gpu_velocity_grid.hpp"
#include "culock.hpp"

__global__ void kernel_print_blocks(GPU_velocity_grid grid) {
    unsigned int tid = blockIdx.x;
    unsigned int ind;
    vel_block_indices_t indices;
    ind = grid.velocity_block_list[tid];
    indices = grid.get_velocity_block_indices(ind);
    printf("%5.0u: (%4i, %4i, %4i) %7.1f\n", ind, indices.ind[0], indices.ind[1], indices.ind[2], grid.block_data[tid*WID3]);
}

// Same as SpatialCell::get_velocity_block_indices but revised for GPU. This must stay in this file so that both host and device versions can be created.
__host__ __device__ vel_block_indices_t GPU_velocity_grid::get_velocity_block_indices(const unsigned int blockid) {
    vel_block_indices_t indices;
    indices.ind[0] = blockid % *vx_length;
    indices.ind[1] = (blockid / *(vx_length)) % *vy_length;
    indices.ind[2] = blockid / (*vx_length * *vy_length);

    return indices;
}

// Wrapper for the kernel
__host__ void GPU_velocity_grid::k_print_blocks(void) {
    kernel_print_blocks<<<*num_blocks, 1>>>(*this);
    CUDACALL(hipPeekAtLastError()); // Check for kernel launch errors
    CUDACALL(hipDeviceSynchronize()); // Check for other cuda errors
}


// Finds the smallest indices in all three dimensions. Returned via (a separately allocated) result array.
__global__ void min_inds(GPU_velocity_grid grid, Lock lock, float *result){
    extern __shared__ float temp[];
    unsigned int *inds = grid.velocity_block_list;
    int tid=threadIdx.x;
    temp[tid]=inds[tid+blockIdx.x*blockDim.x]; // Read data in
    int d;
    for (d=blockDim.x>>1;d>=1;d>>=1){
        __syncthreads();
        if (tid<d) temp[tid] = fmin(temp[tid], temp[tid+d]);
    }
    if (tid==0){
        lock.lock();
        *result=fmin(*result, temp[0]); // Make sure the memory write is complete before letting other threads access it
        __threadfence();
        lock.unlock();
    }
}



#include "hip/hip_runtime.h"
#include "gpu_velocity_grid.hpp"

using namespace spatial_cell;

// Constant memory can not be allocated inside class definition, therefore only accessed from this file via accessors if necessary.
// Identical to those of SpatialCell aka. dimensions of velocity space.
__constant__ unsigned int vx_length, \
                 vy_length, \
                 vz_length;

__global__ void print_constants_k(void) {
    printf("vx_length: %u, vy_length: %u, vz_length: %u\n", vx_length, vy_length, vz_length);
}

void print_constants(void) {
    print_constants_k<<<1,1>>>();
}

// Copies velocity_block_list and block_data as well as necessary constants from a SpatialCell to GPU for processing.
GPU_velocity_grid::GPU_velocity_grid(SpatialCell *spacell) {
	
    // Allocate memory on the gpu
	unsigned int vel_block_list_size = spacell->number_of_blocks*sizeof(unsigned int);
	unsigned int block_data_size = spacell->block_data.size()*sizeof(float);
    hipMallocManaged(&num_blocks, sizeof(unsigned int));
	hipMallocManaged(&velocity_block_list, vel_block_list_size);
	hipMallocManaged(&block_data, block_data_size);
	
	// Copy to gpu
	unsigned int *velocity_block_list_arr = &(spacell->velocity_block_list[0]);
	float *block_data_arr = &(spacell->block_data[0]);
	memcpy(num_blocks, &(spacell->number_of_blocks), sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vx_length), &SpatialCell::vx_length, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vy_length), &SpatialCell::vy_length, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vz_length), &SpatialCell::vz_length, sizeof(unsigned int));
	memcpy(velocity_block_list, velocity_block_list_arr, vel_block_list_size);
	memcpy(block_data, block_data_arr, block_data_size);
}

GPU_velocity_grid::~GPU_velocity_grid() {
    // Free memory
    hipFree(num_blocks);
	hipFree(velocity_block_list);
	hipFree(block_data);
}

// Simple accessors
__device__ inline int GPU_velocity_grid::vx_len(void) {return vx_length;}
__device__ inline int GPU_velocity_grid::vy_len(void) {return vy_length;}
__device__ inline int GPU_velocity_grid::vz_len(void) {return vz_length;}

// Same as SpatialCell::get_velocity_block_indices but revised for GPU. Constructs 3d indices from 1d index.
__device__ ind3d GPU_velocity_grid::get_velocity_block_indices(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % vx_length;
    indices.y = (blockid / vx_length) % vy_length;
    indices.z = blockid / (vx_length * vy_length);

    return indices;
}

// Host version. Requires initialized SpatialCell static variables.
__host__ ind3d GPU_velocity_grid::get_velocity_block_indices_host(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % SpatialCell::vx_length;
    indices.y = (blockid / SpatialCell::vx_length) % SpatialCell::vy_length;
    indices.z = blockid / (SpatialCell::vx_length * SpatialCell::vy_length);

    return indices;
}

// Constructs 1d index out of 3d indices
__device__ unsigned int GPU_velocity_grid::get_velocity_block(const ind3d indices) {
    return indices.x + indices.y * vx_length + indices.z * vx_length * vy_length;
}


// Same as print_blocks, but prints from a kernel
__global__ void kernel_print_blocks(GPU_velocity_grid grid) {
    unsigned int tid = blockIdx.x;
    unsigned int ind;
    ind3d indices;
    ind = grid.velocity_block_list[tid];
    indices = GPU_velocity_grid::get_velocity_block_indices(ind);
    printf("%5.0u: (%4i, %4i, %4i) %7.1f\n", ind, indices.x, indices.y, indices.z, grid.block_data[tid*WID3]);
}

// Wrapper for the kernel
__host__ void GPU_velocity_grid::k_print_blocks(void) {
    kernel_print_blocks<<<*num_blocks, 1>>>(*this);
    CUDACALL(hipPeekAtLastError()); // Check for kernel launch errors
    CUDACALL(hipDeviceSynchronize()); // Check for other cuda errors
}

__host__ void GPU_velocity_grid::print_blocks(void) {
    printf("Number of blocks: %4u.\n", *num_blocks);
    unsigned int ind;
    ind3d indices;
    for (int i=0; i<*num_blocks; i++) {
        ind = velocity_block_list[i];
        printf("%5.0u: ", ind);
        indices = get_velocity_block_indices_host(ind);
        printf("(%4i, %4i, %4i) %7.1f\n", indices.x, indices.y, indices.z, block_data[i*WID3]);
    }
}

#include "hip/hip_runtime.h"
#include "gpu_velocity_grid.hpp"

using namespace spatial_cell;

// Constant memory can not be allocated inside class definition, therefore only accessed directly from this file and via accessors if necessary.
// Identical to those of SpatialCell aka. dimensions of velocity space.
__constant__ unsigned int vx_length, \
                          vy_length, \
                          vz_length;
__constant__ ind3d min3d, max3d;
__constant__ unsigned int min1d, max1d;

// Call with only 1 thread
__global__ void print_constants_k(void) {
    printf("vx_length: %u, vy_length: %u, vz_length: %u\n", vx_length, vy_length, vz_length);
}

void print_constants(void) {
    // Easiest to print from a kernel
    print_constants_k<<<1,1>>>();
}

// Copies velocity_block_list and block_data as well as necessary constants from a SpatialCell to GPU for processing.
GPU_velocity_grid::GPU_velocity_grid(SpatialCell *spacell) {
	
    // Allocate memory on the gpu
	unsigned int vel_block_list_size = spacell->number_of_blocks*sizeof(unsigned int);
	unsigned int block_data_size = spacell->block_data.size()*sizeof(float);
    hipMallocManaged(&num_blocks, sizeof(unsigned int));
	hipMallocManaged(&velocity_block_list, vel_block_list_size);
	hipMallocManaged(&block_data, block_data_size);
	
	// Copy to gpu
	unsigned int *velocity_block_list_arr = &(spacell->velocity_block_list[0]);
	float *block_data_arr = &(spacell->block_data[0]);
	memcpy(num_blocks, &(spacell->number_of_blocks), sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vx_length), &SpatialCell::vx_length, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vy_length), &SpatialCell::vy_length, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vz_length), &SpatialCell::vz_length, sizeof(unsigned int));
	memcpy(velocity_block_list, velocity_block_list_arr, vel_block_list_size);
	memcpy(block_data, block_data_arr, block_data_size);
}

GPU_velocity_grid::~GPU_velocity_grid() {
    // Free memory
    hipFree(num_blocks);
	hipFree(velocity_block_list);
	hipFree(block_data);
	hipFree(vel_grid);
}

// Simple accessors
__device__ inline unsigned int GPU_velocity_grid::vx_len(void) {return vx_length;}
__device__ inline unsigned int GPU_velocity_grid::vy_len(void) {return vy_length;}
__device__ inline unsigned int GPU_velocity_grid::vz_len(void) {return vz_length;}

// Same as SpatialCell::get_velocity_block_indices but revised for GPU. Constructs 3d indices from 1d index.
__device__ ind3d GPU_velocity_grid::get_velocity_block_indices(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % vx_length;
    indices.y = (blockid / vx_length) % vy_length;
    indices.z = blockid / (vx_length * vy_length);

    return indices;
}

// Host version. Requires initialized SpatialCell static variables.
__host__ ind3d GPU_velocity_grid::get_velocity_block_indices_host(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % SpatialCell::vx_length;
    indices.y = (blockid / SpatialCell::vx_length) % SpatialCell::vy_length;
    indices.z = blockid / (SpatialCell::vx_length * SpatialCell::vy_length);

    return indices;
}

// Constructs 1d index out of 3d indices
__device__ unsigned int GPU_velocity_grid::get_velocity_block(const ind3d indices) {
    return indices.x + indices.y * vx_length + indices.z * vx_length * vy_length;
}


// Same as print_blocks, but prints from a kernel
__global__ void kernel_print_blocks(GPU_velocity_grid grid) {
    unsigned int tid = blockIdx.x;
    unsigned int ind;
    ind3d indices;
    ind = grid.velocity_block_list[tid];
    indices = GPU_velocity_grid::get_velocity_block_indices(ind);
    printf("%5.0u: (%4i, %4i, %4i) %7.1f\n", ind, indices.x, indices.y, indices.z, grid.block_data[tid*WID3]);
}

// Wrapper for the kernel
__host__ void GPU_velocity_grid::k_print_blocks(void) {
    kernel_print_blocks<<<*num_blocks, 1>>>(*this);
    CUDACALL(hipPeekAtLastError()); // Check for kernel launch errors
    CUDACALL(hipDeviceSynchronize()); // Check for other cuda errors
}

// Prints information about transferred blocks from gpu memory
__host__ void GPU_velocity_grid::print_blocks(void) {
    printf("Number of blocks: %4u.\n", *num_blocks);
    unsigned int ind;
    ind3d indices;
    for (int i=0; i<*num_blocks; i++) {
        ind = velocity_block_list[i];
        printf("%5.0u: ", ind);
        indices = get_velocity_block_indices_host(ind);
        printf("(%4i, %4i, %4i) %7.1f\n", indices.x, indices.y, indices.z, block_data[i*WID3]);
    }
}

// Returns the data from a given block and cell id.
__device__ float GPU_velocity_grid::get_velocity_cell(unsigned int blockid, unsigned int cellid) {
    ind3d block_indices = GPU_velocity_grid::get_velocity_block_indices(blockid);
    // Check for out of bounds
    if (block_indices.x > max3d.x ||
        block_indices.y > max3d.y ||
        block_indices.z > max3d.z ||
        block_indices.x < min3d.x ||
        block_indices.y < min3d.y ||
        block_indices.z < min3d.z) return ERROR_CELL;
    vel_block *block = &vel_grid[blockid - min1d];
    return block->data[cellid];
}

// Sets the data in a given block and cell id to val. Returns the old value of the cell.
__device__ float GPU_velocity_grid::set_velocity_cell(unsigned int blockid, unsigned int cellid, float val) {
    ind3d block_indices = GPU_velocity_grid::get_velocity_block_indices(blockid);
    // Check for out of bounds
    if (block_indices.x > max3d.x ||
        block_indices.y > max3d.y ||
        block_indices.z > max3d.z ||
        block_indices.x < min3d.x ||
        block_indices.y < min3d.y ||
        block_indices.z < min3d.z) return ERROR_CELL;
    vel_block *block = &vel_grid[blockid - min1d];
    float old = block->data[cellid];
    block->data[cellid] = val;
    return old;
}

// Sets the data in a given block to that of vals.
__device__ void GPU_velocity_grid::set_velocity_block(unsigned int blockid, float *vals) {
    ind3d block_indices = GPU_velocity_grid::get_velocity_block_indices(blockid);
    // Check for out of bounds
    if (block_indices.x > max3d.x ||
        block_indices.y > max3d.y ||
        block_indices.z > max3d.z ||
        block_indices.x < min3d.x ||
        block_indices.y < min3d.y ||
        block_indices.z < min3d.z) return;
    vel_block *block = &vel_grid[blockid - min1d];
    for (int i = 0; i < WID3; i++){
        block->data[i] = vals[i];
    }
    return;
}

// Fills the given array of size len with val
__global__ void init_data(vel_block *grid, float val, int len) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
        for (int j = 0; j < WID3; j++) grid[i].data[j] = val;
    }
    __syncthreads();
}

// Copies data from block_data to vel_grid
__global__ void copy_block_data(GPU_velocity_grid *ggrid) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < *(ggrid->num_blocks)) {
        int blockid = ggrid->velocity_block_list[i];
        //ggrid->set_velocity_block(blockid, &(ggrid->block_data[i*WID3]));
    }
}

// Allocates a full velocity grid and copies data from block_data.
__host__ void GPU_velocity_grid::init_grid(void) {
    unsigned int min = this->min_ind();
    unsigned int max = this->max_ind();
    ind3d min_i = get_velocity_block_indices_host(min);
    ind3d max_i = get_velocity_block_indices_host(max);
    
    // dimensions of the grid
    unsigned int dx = max_i.x - min_i.x;
    unsigned int dy = max_i.y - min_i.y;
    unsigned int dz = max_i.z - min_i.z;
    unsigned int grid_len = dx*dy*dz;
    
    // Copy to constant memory
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(min1d), &min, sizeof(unsigned int)));
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(max1d), &max, sizeof(unsigned int)));
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(min3d), &min_i, sizeof(ind3d)));
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(max3d), &max_i, sizeof(ind3d)));
    CUDACALL(hipMalloc(&vel_grid, grid_len * sizeof(vel_block)));
    
    // Calculate grid dimensions and start kernel
    unsigned int blockSize = 64;
    unsigned int gridSize = ceilDivide(grid_len, blockSize);
    init_data<<<gridSize, blockSize>>>(vel_grid, 0.0f, grid_len);
    hipMemcpy(&gridSize, num_blocks, sizeof(unsigned int), hipMemcpyDeviceToHost);
    gridSize = ceilDivide(gridSize, blockSize);
    copy_block_data<<<gridSize, blockSize>>>(this);
}



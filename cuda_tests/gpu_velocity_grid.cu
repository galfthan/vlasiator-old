#include "hip/hip_runtime.h"
#include "gpu_velocity_grid.hpp"

using namespace spatial_cell;

// Constant memory can not be allocated inside class definition, therefore only accessed directly from this file and via accessors if necessary.
// Identical to those of SpatialCell aka. dimensions of velocity space.
__constant__ unsigned int vx_length, \
                          vy_length, \
                          vz_length;
// Minimum and maximum points of bounding box and lengths of each dimension.
__constant__ ind3d min3d, max3d, box_dims;

// Call with only 1 thread
__global__ void print_constants_k(void) {
    printf("vx_length: %u, vy_length: %u, vz_length: %u\n", vx_length, vy_length, vz_length);
}

void print_constants(void) {
    // Easiest to print from a kernel
    print_constants_k<<<1,1>>>();
}

// Copies velocity_block_list and block_data as well as necessary constants from a SpatialCell to GPU for processing.
GPU_velocity_grid::GPU_velocity_grid(SpatialCell *spacell) {
	
    // Allocate memory on the gpu
	unsigned int vel_block_list_size = spacell->number_of_blocks*sizeof(unsigned int);
	unsigned int block_data_size = spacell->block_data.size()*sizeof(float);
    hipMallocManaged(&num_blocks, sizeof(unsigned int));
	hipMallocManaged(&velocity_block_list, vel_block_list_size);
	hipMallocManaged(&block_data, block_data_size);
	
	// Copy to gpu
	unsigned int *velocity_block_list_arr = &(spacell->velocity_block_list[0]);
	float *block_data_arr = &(spacell->block_data[0]);
	memcpy(num_blocks, &(spacell->number_of_blocks), sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vx_length), &SpatialCell::vx_length, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vy_length), &SpatialCell::vy_length, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(vz_length), &SpatialCell::vz_length, sizeof(unsigned int));
	memcpy(velocity_block_list, velocity_block_list_arr, vel_block_list_size);
	memcpy(block_data, block_data_arr, block_data_size);
}

GPU_velocity_grid::~GPU_velocity_grid() {
    // Free memory
    hipFree(num_blocks);
	hipFree(velocity_block_list);
	hipFree(block_data);
	hipFree(vel_grid);
}

// Simple accessors
__device__ inline unsigned int GPU_velocity_grid::vx_len(void) {return vx_length;}
__device__ inline unsigned int GPU_velocity_grid::vy_len(void) {return vy_length;}
__device__ inline unsigned int GPU_velocity_grid::vz_len(void) {return vz_length;}

// Same as SpatialCell::get_velocity_block_indices but revised for GPU. Constructs 3d indices from 1d index.
__device__ ind3d GPU_velocity_grid::get_velocity_block_indices(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % vx_length;
    indices.y = (blockid / vx_length) % vy_length;
    indices.z = blockid / (vx_length * vy_length);

    return indices;
}

// Host version. Requires initialized SpatialCell static variables.
__host__ ind3d GPU_velocity_grid::get_velocity_block_indices_host(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % SpatialCell::vx_length;
    indices.y = (blockid / SpatialCell::vx_length) % SpatialCell::vy_length;
    indices.z = blockid / (SpatialCell::vx_length * SpatialCell::vy_length);

    return indices;
}

// Constructs 1d index out of 3d indices
__device__ unsigned int GPU_velocity_grid::get_velocity_block(const ind3d indices) {
    return indices.x + indices.y * vx_length + indices.z * vx_length * vy_length;
}


// Same as print_blocks, but prints from a kernel
__global__ void kernel_print_blocks(GPU_velocity_grid grid) {
    unsigned int tid = blockIdx.x;
    unsigned int ind;
    ind3d indices;
    ind = grid.velocity_block_list[tid];
    indices = GPU_velocity_grid::get_velocity_block_indices(ind);
    printf("%5.0u: (%4i, %4i, %4i) %7.1f\n", ind, indices.x, indices.y, indices.z, grid.block_data[tid*WID3]);
}

// Wrapper for the kernel
__host__ void GPU_velocity_grid::k_print_blocks(void) {
    kernel_print_blocks<<<*num_blocks, 1>>>(*this);
    CUDACALL(hipPeekAtLastError()); // Check for kernel launch errors
    CUDACALL(hipDeviceSynchronize()); // Check for other cuda errors
}

// Prints information about transferred blocks from gpu memory
__host__ void GPU_velocity_grid::print_blocks(void) {
    printf("Number of blocks: %4u.\n", *num_blocks);
    unsigned int ind;
    ind3d indices;
    for (int i=0; i<*num_blocks; i++) {
        ind = velocity_block_list[i];
        printf("%5.0u: ", ind);
        indices = get_velocity_block_indices_host(ind);
        printf("(%4i, %4i, %4i) %7.1f\n", indices.x, indices.y, indices.z, block_data[i*WID3]);
    }
}

__device__ vel_block* GPU_velocity_grid::get_velocity_grid_block(unsigned int blockid) {
    ind3d block_indices = GPU_velocity_grid::get_velocity_block_indices(blockid);
    // Check for out of bounds
    if (block_indices.x > max3d.x ||
        block_indices.y > max3d.y ||
        block_indices.z > max3d.z ||
        block_indices.x < min3d.x ||
        block_indices.y < min3d.y ||
        block_indices.z < min3d.z) return ERROR_BLOCK;
    // Move the indices to same origin and dimensions as the bounding box
    ind3d n_ind = {block_indices.x - min3d.x, block_indices.y - min3d.y, block_indices.z - min3d.z};
    vel_block *block_ptr = &vel_grid[n_ind.x + n_ind.y*box_dims.x + n_ind.z*box_dims.x*box_dims.y];
    return block_ptr;
}

// Returns index of the full grid corresponding to the blockid of the sparse grid
__device__ int GPU_velocity_grid::get_velocity_grid_block_ind(unsigned int blockid) {
    ind3d block_indices = GPU_velocity_grid::get_velocity_block_indices(blockid);
    // Check for out of bounds
    if (block_indices.x > max3d.x ||
        block_indices.y > max3d.y ||
        block_indices.z > max3d.z ||
        block_indices.x < min3d.x ||
        block_indices.y < min3d.y ||
        block_indices.z < min3d.z) return -1;
    // Move the indices to same origin and dimensions as the bounding box
    ind3d n_ind = {block_indices.x - min3d.x, block_indices.y - min3d.y, block_indices.z - min3d.z};
    unsigned int ind = n_ind.x + n_ind.y*box_dims.x + n_ind.z*box_dims.x*box_dims.y;
    return ind;    
}

// Returns the data from a given block and cell id.
__device__ float GPU_velocity_grid::get_velocity_cell(unsigned int blockid, unsigned int cellid) {
    vel_block *block = get_velocity_grid_block(blockid);
    // Check for out of bounds
    if (block == ERROR_BLOCK) return ERROR_CELL;
    return block->data[cellid];
}

// Sets the data in a given block and cell id to val. Returns the old value of the cell.
__device__ float GPU_velocity_grid::set_velocity_cell(unsigned int blockid, unsigned int cellid, float val) {
    vel_block *block = get_velocity_grid_block(blockid);
    // Check for out of bounds
    if (block == ERROR_BLOCK) return ERROR_CELL;
    float old = block->data[cellid];
    block->data[cellid] = val;
    return old;
}

// Sets the data in a given block to that of vals.
__device__ void GPU_velocity_grid::set_velocity_block(unsigned int blockid, float *vals) {
    vel_block *block = get_velocity_grid_block(blockid);
    // Check for out of bounds
    if (block == ERROR_BLOCK) return;
    for (int i = 0; i < WID3; i++){
        block->data[i] = vals[i];
    }
    __syncthreads();
    return;
}

// Fills the given array of size len with val
__global__ void init_data(vel_block *grid, float val, int len) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
        for (int j = 0; j < WID3; j++) {
            grid[i].data[j] = val;
        }
    }
}

// Copies data from block_data to vel_grid
__global__ void copy_block_data(GPU_velocity_grid ggrid) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("%i %i\n", i, *(ggrid.num_blocks));
    if (i < *(ggrid.num_blocks)) {
        int blockid = ggrid.velocity_block_list[i];
        ggrid.set_velocity_block(blockid, &(ggrid.block_data[i*WID3]));
    }
}

// Allocates a full velocity grid and copies data from block_data.
__host__ void GPU_velocity_grid::init_grid(void) {
    unsigned int min = this->min_ind();
    unsigned int max = this->max_ind();
    ind3d min_i = get_velocity_block_indices_host(min);
    ind3d max_i = get_velocity_block_indices_host(max);
    printf("MIN: %u %u %u %u\n", min, min_i.x, min_i.y, min_i.z);
    printf("MAX: %u %u %u %u\n", max, max_i.x, max_i.y, max_i.z);
    // dimensions of the grid
    unsigned int dx = max_i.x - min_i.x + 1;
    unsigned int dy = max_i.y - min_i.y + 1;
    unsigned int dz = max_i.z - min_i.z + 1;
    unsigned int vel_grid_len = dx*dy*dz;
    printf("GRID DIMS: %u %u %u\n", dx, dy, dz);
    ind3d dims = {dx, dy, dz};
    // Copy to constant memory
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(min3d), &min_i, sizeof(ind3d)));
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(max3d), &max_i, sizeof(ind3d)));
    CUDACALL(hipMemcpyToSymbol(HIP_SYMBOL(box_dims), &dims, sizeof(ind3d)));
    CUDACALL(hipMalloc(&vel_grid, vel_grid_len * sizeof(vel_block)));
    
    // Calculate grid dimensions and start kernel
    unsigned int blockSize = 64;
    unsigned int gridSize = ceilDivide(vel_grid_len, blockSize);
    init_data<<<gridSize, blockSize>>>(vel_grid, 0.0f, vel_grid_len);
    CUDACALL(hipMemcpy(&gridSize, num_blocks, sizeof(unsigned int), hipMemcpyDeviceToHost));
    printf("%u ", vel_grid_len);
    printf("%u %u\n", gridSize, blockSize);
    gridSize = ceilDivide(gridSize, blockSize);
    CUDACALL(hipDeviceSynchronize()); // Wait for initialization to finish
    copy_block_data<<<gridSize, blockSize>>>(*this);
    CUDACALL(hipDeviceSynchronize()); // Block before returning
}



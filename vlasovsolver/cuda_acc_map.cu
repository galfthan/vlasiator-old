#include "../velocity_mesh_cuda.h"


bool accelerateVelocityMeshCuda(Realf **blockDatas, vmesh::GlobalID **blockIDs, vmesh::LocalID *nBlocks, const uint nCells ){
//   hipStream_t streams[nCells];   
   for (int i = 0; i < nCells; i++) {
     // TODO, put each cell into a separate stream:      hipStreamCreate(&streams[i]);
      vmesh::VelocityMeshCuda<vmesh::GlobalID, vmesh::LocalID> *d_vmesh =
         vmesh::createVelocityMeshCuda(blockDatas[i], blockIDs[i], nBlocks[i]);

      vmesh::destroyVelocityMeshCuda(d_vmesh);
   }
   
   hipDeviceSynchronize();
}
